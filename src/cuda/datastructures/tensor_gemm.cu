#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "datastructures/tensor_operations.hpp"

template <typename T>
__global__ void gemmKernel(int M, int N, int K, T ALPHA, T BETA, const T* A,
                           int lda, const T* B, int ldb, T* C, int ldc) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < M && col < N) {
    T sum = 0;
    for (int i = 0; i < K; ++i) sum += A[row * lda + i] * B[i * ldb + col];
    C[row * ldc + col] = BETA * C[row * ldc + col] + ALPHA * sum;
  }
}

#define CUDA_CHECK(expr)                                                       \
  do {                                                                         \
    hipError_t err = (expr);                                                  \
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err)); \
  } while (0)

template <typename T>
void TensorOperations<T>::gemm(int TA, int TB, int M, int N, int K, T ALPHA,
                               T BETA, std::shared_ptr<Tensor<T>> A, int lda,
                               std::shared_ptr<Tensor<T>> B, int ldb,
                               std::shared_ptr<Tensor<T>> C, int ldc) {
  const auto A_shape = A->get_shape();
  const auto B_shape = B->get_shape();
  const auto C_shape = C->get_shape();

  if (C_shape[0] != M || C_shape[1] != N) {
    throw std::invalid_argument("Output matrix C dimensions don't match M×N");
  }

  if (!TA && !TB) {
    if (A_shape[0] != M || A_shape[1] != K || B_shape[0] != K ||
        B_shape[1] != N) {
      throw std::invalid_argument(
          "Input matrix dimensions don't match for multiplication");
    }
  } else if (TA && !TB) {
    if (A_shape[0] != K || A_shape[1] != M || B_shape[0] != K ||
        B_shape[1] != N) {
      throw std::invalid_argument(
          "Input matrix dimensions don't match for multiplication with A "
          "transposed");
    }
  } else if (!TA && TB) {
    if (A_shape[0] != M || A_shape[1] != K || B_shape[0] != N ||
        B_shape[1] != K) {
      throw std::invalid_argument(
          "Input matrix dimensions don't match for multiplication with B "
          "transposed");
    }
  } else {
    if (A_shape[0] != K || A_shape[1] != M || B_shape[0] != N ||
        B_shape[1] != K) {
      throw std::invalid_argument(
          "Input matrix dimensions don't match for multiplication with both "
          "matrices transposed");
    }
  }

  if (TA) A = A->transpose();
  if (TB) B = B->transpose();

  const T* hA = A->get_data().get();
  const T* hB = B->get_data().get();
  T* hC = const_cast<T*>(C->get_data().get());

  T *dA = nullptr, *dB = nullptr, *dC = nullptr;
  size_t sizeA = sizeof(T) * size_t(lda) * size_t(M);
  size_t sizeB = sizeof(T) * size_t(ldb) * size_t(K);
  size_t sizeC = sizeof(T) * size_t(ldc) * size_t(M);
  CUDA_CHECK(hipMalloc(&dA, sizeA));
  CUDA_CHECK(hipMalloc(&dB, sizeB));
  CUDA_CHECK(hipMalloc(&dC, sizeC));

  CUDA_CHECK(hipMemcpy(dA, hA, sizeA, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dB, hB, sizeB, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dC, hC, sizeC, hipMemcpyHostToDevice));

  constexpr int TILE = 16;
  dim3 block(TILE, TILE);
  dim3 grid((N + TILE - 1) / TILE, (M + TILE - 1) / TILE);
  gemmKernel<T>
      <<<grid, block>>>(M, N, K, ALPHA, BETA, dA, lda, dB, ldb, dC, ldc);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipMemcpy(hC, dC, sizeC, hipMemcpyDeviceToHost));

  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
}

#define TYPE(DT) _TENSOR_OPERATIONS(DT)
#include "types_integer.txt"
#include "types_real.txt"
#undef TYPE